#include <iostream>
#include <hip/hip_runtime.h>

int main(int argc, char** argv) {
    int deviceCount;

    hipGetDeviceCount(&deviceCount);
    std::cout << "Device Count: " << deviceCount << std::endl;

    for (int device = 0; device < deviceCount; device++) {
        hipDeviceProp_t deviceProp; 

        hipGetDeviceProperties(&deviceProp, device);

        if (device == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
                std::cout << "No CUDA GPU has been detected" << std::endl;
            } else if (deviceCount == 1) {
                std::cout << "There is one device supporting CUDA" << std::endl;
            } else {
                std::cout << "There are multiple devices supporting CUDA" << std::endl;
            }
        }

        std::cout << "Device: " << device << ", name: " << deviceProp.name << std::endl;
        std::cout << "Computational Capabilities: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << "Maximum global memory size: " << deviceProp.totalGlobalMem << std::endl;
        std::cout << "Maximum constant memory size: " << deviceProp.totalConstMem << std::endl;
        std::cout << "Maximum shared memory size: " << deviceProp.sharedMemPerBlock << std::endl;
        // Max block dim tells us how many threads per block in x and y dim and then last num is threads in z dim
        std::cout << "Maximum block dimensions: " << deviceProp.maxThreadsDim[0] << " x " << deviceProp.maxThreadsDim[1] << " x " << deviceProp.maxThreadsDim[2] << std::endl;
        std::cout << "Maximum grid dimensions: " << deviceProp.maxGridSize[0] << " x " << deviceProp.maxGridSize[1] << " x " << deviceProp.maxGridSize[2] << std::endl;
        // warp size is how many threads can be executed at the same time
        std::cout << "Warp size: " << deviceProp.warpSize << std::endl;
    }
    return 0;
}
