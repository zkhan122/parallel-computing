
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

/*
remember the kernel is the device code and is program that 
controls the hardware
*/

// adding elements from 2 arrays and finding max sum

// add function definition

// void add(int n, float *x, float *y) { // host code

// now we are running on the GPU instead (CUDA Kernel function) - device code
__global__ 
void add(int n, float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    int N = 1<<20; // 1M elements

    float *x, *y;

    // allocate unified memory accessible from the CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));


    // initialize x and y arrays on the host 
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the CPU
    // add(N, x, y);

    // Running on GPU CUDA kernel
    add<<<1, 1>>>(N, x, y);

    // make sure the GPU completes its threads first
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    }
    std::cout << "Max Error: " << maxError << std::endl;

    // Free memory
    // delete [] x;
    // delete [] y;

    // freeing memory using CUDA
    hipFree(x);
    hipFree(y);

    return 0;
}